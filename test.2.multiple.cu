// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cmath>

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Randomization helpers 
// adapted from https://github.com/ROCmSoftwarePlatform/rocBLAS/blob/rocm-3.0/clients/include/rocblas_init.hpp#L42

void fill_sin(float *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = sin(float(i + j * nr_rows_A));
}


void fill_cos(float *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = cos(float(i + j * nr_rows_A));
}


#ifdef CUBLAS_API_H_
// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
	switch (error)
	{
		case HIPBLAS_STATUS_SUCCESS:
			return "CUBLAS_STATUS_SUCCESS";

		case HIPBLAS_STATUS_NOT_INITIALIZED:
			return "CUBLAS_STATUS_NOT_INITIALIZED";

		case HIPBLAS_STATUS_ALLOC_FAILED:
			return "CUBLAS_STATUS_ALLOC_FAILED";

		case HIPBLAS_STATUS_INVALID_VALUE:
			return "CUBLAS_STATUS_INVALID_VALUE";

		case HIPBLAS_STATUS_ARCH_MISMATCH:
			return "CUBLAS_STATUS_ARCH_MISMATCH";

		case HIPBLAS_STATUS_MAPPING_ERROR:
			return "CUBLAS_STATUS_MAPPING_ERROR";

		case HIPBLAS_STATUS_EXECUTION_FAILED:
			return "CUBLAS_STATUS_EXECUTION_FAILED";

		case HIPBLAS_STATUS_INTERNAL_ERROR:
			return "CUBLAS_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}
#endif

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
	hipError_t e=hipGetLastError();                                 \
	if(e!=hipSuccess) {                                              \
		printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
		exit(0); \
	}                                                                 \
}


void setup_nvlink(int numGPUs){
	if (numGPUs >= 4){
		numGPUs = 3;
	}

	for (int i = 0; i <= numGPUs; i++) {
		for (int j = i+1; j <= numGPUs; j++) {
		int access = 0;
		hipDeviceCanAccessPeer(&access, i, j);
		if (access) {
			printf("Enabling %d to %d\n", i, j);
			hipSetDevice(i);
			cudaCheckError();
			hipDeviceEnablePeerAccess(j, 0);
			cudaCheckError();
			hipSetDevice(j);
			cudaCheckError();
			hipDeviceEnablePeerAccess(i, 0);
			cudaCheckError();
			hipSetDevice(i);
			cudaCheckError();
		}
		fflush(stdout);
		}
	}
}


// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul( hipblasHandle_t handle, const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	hipblasStatus_t err = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	if (err != HIPBLAS_STATUS_SUCCESS)
		std::cout << "Error: " <<  _cudaGetErrorEnum(err) << std::endl;

}


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	for(int i = 0; i < nr_rows_A; ++i){
		for(int j = 0; j < nr_cols_A; ++j){
			std::cout << A[j * nr_rows_A + i] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

int main(int argc, char* argv[]) {

	if (argc != 4){
		std::cout << "USAGE: " << argv[0] <<" <size> <inner-reps> nlinks" <<std::endl ;
		exit(-1);
	}
	int size = atoi(argv[1]);
	int reps = atoi(argv[2]);
	int nlinks = atoi(argv[3]);

	setup_nvlink(nlinks);

	hipStream_t computeStream;
	hipStream_t compStrm1, compStrm2, compStrm3;
	hipError_t result;

	result = hipStreamCreateWithFlags(&computeStream, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&compStrm1, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&compStrm2, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&compStrm3, hipStreamNonBlocking);

	hipStream_t copyStream, copyStream2;
	hipStream_t copyStream3, copyStream4;
	result = hipStreamCreateWithFlags(&copyStream, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream2, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream3, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream4, hipStreamNonBlocking);



	// Allocate the src on CPU
	long SIZE = 512*1024*1024;
	// int* src = (int*) malloc(SIZE * sizeof(int));
	int* src; 
	int *dest_h; 
	hipHostMalloc((void**) &src, SIZE * sizeof(int), hipHostMallocDefault);
	hipHostMalloc((void**) &dest_h, SIZE * sizeof(int), hipHostMallocDefault);
	for (int i = 0; i < SIZE ; ++i) {
		src[i] = sin(i);
		dest_h[i] = 1;
	}

	hipSetDevice(0);
	// Allocate DST on gpu	
	int* dst;
	hipMalloc(&dst, SIZE*sizeof(int));

	// Allocate buffers on all cpus:
	hipSetDevice(1);
	int* src_1;
	hipMalloc(&src_1, SIZE*sizeof(int));

	hipMemcpy((void*)src_1, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipSetDevice(2);
	int* src_2;
	hipMalloc(&src_2, SIZE*sizeof(int));

	hipMemcpy((void*)src_2, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipSetDevice(3);
	int* src_3;
	hipMalloc(&src_3, SIZE*sizeof(int));

	hipMemcpy((void*)src_3, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();


	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	// for simplicity we are going to use square arrays
	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = size;

	float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
	float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
	float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));
	
	// Fill the arrays A and B on GPU with random numbers
	fill_sin(h_A, nr_rows_A, nr_cols_A);
	fill_cos(h_B, nr_rows_B, nr_cols_B);
	
	hipSetDevice(0);

	hipDeviceSynchronize();
	// Allocate 3 arrays on GPU
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
	hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
	hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));
	// copy data to device
	hipMemcpyAsync(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice, computeStream);
	hipMemcpyAsync(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice, computeStream);
	hipMemcpyAsync(d_C,h_A,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice, computeStream);
	hipDeviceSynchronize();

	hipSetDevice(1);
	hipDeviceSynchronize();
	// Allocate 3 arrays on GPU
	float *d_A_1, *d_B_1, *d_C_1;
	hipMalloc(&d_A_1,nr_rows_A * nr_cols_A * sizeof(float));
	hipMalloc(&d_B_1,nr_rows_B * nr_cols_B * sizeof(float));
	hipMalloc(&d_C_1,nr_rows_C * nr_cols_C * sizeof(float));
	// copy data to device
	hipMemcpyAsync(d_A_1,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice, compStrm1);
	hipMemcpyAsync(d_B_1,h_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice, compStrm1);
	hipMemcpyAsync(d_C_1,h_A,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice, compStrm1);
	
	hipDeviceSynchronize();

	hipSetDevice(2);
	hipDeviceSynchronize();
	// Allocate 3 arrays on GPU
	float *d_A_2, *d_B_2, *d_C_2;
	hipMalloc(&d_A_2,nr_rows_A * nr_cols_A * sizeof(float));
	hipMalloc(&d_B_2,nr_rows_B * nr_cols_B * sizeof(float));
	hipMalloc(&d_C_2,nr_rows_C * nr_cols_C * sizeof(float));
	// copy data to device
	hipMemcpyAsync(d_A_2,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice, compStrm2);
	hipMemcpyAsync(d_B_2,h_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice, compStrm2);
	hipMemcpyAsync(d_C_2,h_A,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice, compStrm2);
	
	hipDeviceSynchronize();

	hipSetDevice(3);

	hipDeviceSynchronize();
	// Allocate 3 arrays on GPU
	float *d_A_3, *d_B_3, *d_C_3;
	hipMalloc(&d_A_3,nr_rows_A * nr_cols_A * sizeof(float));
	hipMalloc(&d_B_3,nr_rows_B * nr_cols_B * sizeof(float));
	hipMalloc(&d_C_3,nr_rows_C * nr_cols_C * sizeof(float));
	// copy data to device
	hipMemcpyAsync(d_A_3,h_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyHostToDevice, compStrm3);
	hipMemcpyAsync(d_B_3,h_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice, compStrm3);
	hipMemcpyAsync(d_C_3,h_A,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyHostToDevice, compStrm3);
	
	hipDeviceSynchronize();

	std::cout << "A =" << std::endl;
	// print_matrix(h_A, nr_rows_A, nr_cols_A);
	std::cout << "B =" << std::endl;
	// print_matrix(h_B, nr_rows_B, nr_cols_B);
	hipMemcpyAsync((void*)dst, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice, copyStream);
	
	hipSetDevice(0);
	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetStream(handle, computeStream);
	hipDeviceSynchronize();

	hipSetDevice(1);
	hipblasHandle_t handle1;
	hipblasCreate(&handle1);
	hipblasSetStream(handle, compStrm1);
	hipDeviceSynchronize();

	hipSetDevice(2);
	hipblasHandle_t handle2;
	hipblasCreate(&handle2);
	hipblasSetStream(handle, compStrm2);
	hipDeviceSynchronize();

	hipSetDevice(3);
	hipblasHandle_t handle3;
	hipblasCreate(&handle3);
	hipblasSetStream(handle, compStrm3);
	hipDeviceSynchronize();

	hipSetDevice(0);

	for (int j = 0 ; j < 100; j++){
		for (int gpu=0; gpu <= nlinks; gpu++){ 
			switch(gpu){
				case 0:
					// Takes about 5 minuets
					hipSetDevice(0);
					gpu_blas_mmul(handle, d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
					for (int i=0; i< reps; i++){
						// each stable copy takes about 162 miliseconds
						// cudaMemcpyAsync((void*)src, (void*)dst, sizeof(int) * SIZE, cudaMemcpyDeviceToHost, copyStream);
						// cudaMemcpyAsync((void*)dest_h, (void*)dst, sizeof(int) * SIZE, cudaMemcpyDeviceToHost, copyStream2);
						if (nlinks >=1)  
							hipMemcpyAsync((void*)src_1, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToDevice, copyStream2);
						if (nlinks >= 2)
							hipMemcpyAsync((void*)src_2, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToDevice, copyStream3);
						if (nlinks >= 3)
							hipMemcpyAsync((void*)src_3, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToDevice, copyStream4);

					}
					break;
				
				case 1:
					hipSetDevice(1);
					gpu_blas_mmul(handle1, d_A_1, d_B_1, d_C_1, nr_rows_A, nr_cols_A, nr_cols_B);
					break;
				
				case 2:
					hipSetDevice(2);
					gpu_blas_mmul(handle2, d_A_2, d_B_2, d_C_2, nr_rows_A, nr_cols_A, nr_cols_B);
					break;
				case 3:
					hipSetDevice(3);
					gpu_blas_mmul(handle3, d_A_3, d_B_3, d_C_3, nr_rows_A, nr_cols_A, nr_cols_B);
					break;
			}
			// Create a handle for CUBLAS
		}
		hipSetDevice(0);
		hipStreamSynchronize(copyStream);
		hipStreamSynchronize(copyStream2);
		hipStreamSynchronize(copyStream3);
		hipStreamSynchronize(copyStream4);

	}

	hipStreamSynchronize(computeStream);
	hipStreamSynchronize(copyStream);
	hipStreamSynchronize(copyStream2);
	hipStreamSynchronize(copyStream3);
	hipStreamSynchronize(copyStream4);
	
	hipSetDevice(1);
	hipStreamSynchronize(compStrm1);
	hipSetDevice(2);
	hipStreamSynchronize(compStrm2);
	hipSetDevice(3);
	hipStreamSynchronize(compStrm3);  
	
	hipSetDevice(0);
	hipDeviceSynchronize();
	// Destroy the handle
	hipblasDestroy(handle);


	// Copy (and print) the result on host memory
	hipMemcpyAsync(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost, computeStream);
	std::cout << "C =" << std::endl;
	// print_matrix(h_C, nr_rows_C, nr_cols_C);

	// cudaMemcpyAsync(h_C,d_C_1,nr_rows_C * nr_cols_C * sizeof(float),cudaMemcpyDeviceToHost, compStrm1);
	// std::cout << "C =" << std::endl;

	// cudaMemcpyAsync(h_C,d_C_2,nr_rows_C * nr_cols_C * sizeof(float),cudaMemcpyDeviceToHost, compStrm1);
	// std::cout << "C =" << std::endl;

	// cudaMemcpyAsync(h_C,d_C_3,nr_rows_C * nr_cols_C * sizeof(float),cudaMemcpyDeviceToHost, compStrm1);
	// std::cout << "C =" << std::endl;

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	hipHostFree(src);
	hipHostFree(dest_h);
	hipFree(dst);	

	hipSetDevice(1);
	hipFree(src_1);
	hipFree(d_A_1);
	hipFree(d_B_1);
	hipFree(d_C_1);
	hipblasDestroy(handle1);
	result = hipStreamDestroy(compStrm1);



	hipSetDevice(2);
	hipFree(src_2);
	hipFree(d_A_2);
	hipFree(d_B_2);
	hipFree(d_C_2);
	hipblasDestroy(handle2);
	result = hipStreamDestroy(compStrm2);


	hipSetDevice(3);
	hipFree(src_3);
	hipFree(d_A_3);
	hipFree(d_B_3);
	hipFree(d_C_3);
	hipblasDestroy(handle3);
	result = hipStreamDestroy(compStrm3);

	hipSetDevice(0);
	result = hipStreamDestroy(computeStream);
	result = hipStreamDestroy(copyStream);
	result = hipStreamDestroy(copyStream2);
	result = hipStreamDestroy(copyStream3);
	result = hipStreamDestroy(copyStream4);

	// Free CPU memory
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
