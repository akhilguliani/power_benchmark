#include "hip/hip_runtime.h"
// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	for(int i = 0; i < nr_rows_A; ++i){
		for(int j = 0; j < nr_cols_A; ++j){
			std::cout << A[j * nr_rows_A + i] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}


void setup_nvlink(gpu){
	int numGPUs = 4;

	int i = gpu;
	for (int j = 0; j < numGPUs; j++) {
		int access = 0;
		hipDeviceCanAccessPeer(&access, i, j);
		if (access) {
			printf("Enabling %d to %d\n", i, j);
			hipSetDevice(i);
			cudaCheckError();
			hipDeviceEnablePeerAccess(j, 0);
			cudaCheckError();
			hipSetDevice(j);
			cudaCheckError();
			hipDeviceEnablePeerAccess(i, 0);
			cudaCheckError();
			hipSetDevice(i);
			cudaCheckError();
		}
		fflush(stdout);
	}
}

void fill_sin(float *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = sin(float(i + j * nr_rows_A));
}

int main(int argc, char* argv[]) {
        
	if (argc != 8){
		std::cout << "USAGE: " << argv[0] <<"<size> <reps> <active-links>" <<std::endl ;
		exit(-1);
	}

	int multiplier = atoi(argv[1]);

	int reps = atoi(argv[2]);
	
	int nlinks = atoi(argv[3]);

	int gpu = atoi(argv[4]);
	int d1 = atoi(argv[5]);
	int d2 = atoi(argv[6]);
	int d3 = atoi(argv[7]);
        
    setup_nvlink(gpu);

	hipSetDevice(gpu);
	hipStream_t copyStream, copyStream2;
	hipStream_t copyStream3, copyStream4;
	hipError_t result = hipStreamCreateWithFlags(&copyStream, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream2, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream3, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream4, hipStreamNonBlocking);

	// Allocate the src on CPU
	long SIZE = multiplier*1024*1024;
	// int* src = (int*) malloc(SIZE * sizeof(int));
	int* src; 
	hipHostMalloc((void**) &src, SIZE * sizeof(int));

	int* src_h; 
	hipHostMalloc((void**) &src_h, SIZE * sizeof(int));

	for (int i = 0; i < SIZE ; ++i) {
		src[i] = sin(i);
		src_h[i] = 4;
	}

	// Allocate DST on gpu	
	int* dst;
	hipMalloc(&dst, SIZE*sizeof(int));

	hipMemcpyAsync((void*)dst, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice, copyStream);
	
	hipSetDevice(d1);
	int* src_1;
	hipMalloc(&src_1, SIZE*sizeof(int));

	hipMemcpy((void*)src_1, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipSetDevice(d2);
	int* src_2;
	hipMalloc(&src_2, SIZE*sizeof(int));

	hipMemcpy((void*)src_2, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipSetDevice(d3);
	int* src_3;
	hipMalloc(&src_3, SIZE*sizeof(int));

	hipMemcpy((void*)src_3, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();


	hipSetDevice(gpu);

	hipDeviceSynchronize();
	
	for (int i = 0; i < reps; i++) {
	    hipMemcpyAsync((void*)src, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToHost, copyStream);
	    if (nlinks >=1)  
	    	hipMemcpyAsync((void*)src_1, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToHost, copyStream2);
	    if (nlinks >= 2)
	    	hipMemcpyAsync((void*)src_2, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToHost, copyStream3);
	    if (nlinks >= 3)
	    	hipMemcpyAsync((void*)src_3, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToHost, copyStream4);
        }

	// Create a handle for CUBLAS	        
	hipStreamSynchronize(copyStream);
	hipStreamSynchronize(copyStream2);
	hipStreamSynchronize(copyStream3);
	hipStreamSynchronize(copyStream4);

	//Free pinned memory
	hipHostFree(src);
	hipFree(dst);
	
	hipSetDevice(d1);
	hipFree(src_1);
	
	hipSetDevice(d2);
	hipFree(src_2);

	hipSetDevice(d3);
	hipFree(src_3);


	result = hipStreamDestroy(copyStream);
	result = hipStreamDestroy(copyStream2);
	result = hipStreamDestroy(copyStream3);
	result = hipStreamDestroy(copyStream4);
	return 0;
}
