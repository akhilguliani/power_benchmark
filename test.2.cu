// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}


#ifdef CUBLAS_API_H_
// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
	switch (error)
	{
		case HIPBLAS_STATUS_SUCCESS:
			return "CUBLAS_STATUS_SUCCESS";

		case HIPBLAS_STATUS_NOT_INITIALIZED:
			return "CUBLAS_STATUS_NOT_INITIALIZED";

		case HIPBLAS_STATUS_ALLOC_FAILED:
			return "CUBLAS_STATUS_ALLOC_FAILED";

		case HIPBLAS_STATUS_INVALID_VALUE:
			return "CUBLAS_STATUS_INVALID_VALUE";

		case HIPBLAS_STATUS_ARCH_MISMATCH:
			return "CUBLAS_STATUS_ARCH_MISMATCH";

		case HIPBLAS_STATUS_MAPPING_ERROR:
			return "CUBLAS_STATUS_MAPPING_ERROR";

		case HIPBLAS_STATUS_EXECUTION_FAILED:
			return "CUBLAS_STATUS_EXECUTION_FAILED";

		case HIPBLAS_STATUS_INTERNAL_ERROR:
			return "CUBLAS_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}
#endif

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
	hipError_t e=hipGetLastError();                                 \
	if(e!=hipSuccess) {                                              \
		printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
		exit(0); \
	}                                                                 \
}


void setup_nvlink(){
	int numGPUs = 4;

	int i = 0;
	for (int j = 0; j < numGPUs; j++) {
		int access = 0;
		hipDeviceCanAccessPeer(&access, i, j);
		if (access) {
			printf("Enabling %d to %d\n", i, j);
			hipSetDevice(i);
			cudaCheckError();
			hipDeviceEnablePeerAccess(j, 0);
			cudaCheckError();
			hipSetDevice(j);
			cudaCheckError();
			hipDeviceEnablePeerAccess(i, 0);
			cudaCheckError();
			hipSetDevice(i);
			cudaCheckError();
		}
		fflush(stdout);
	}
}


// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul( hipblasHandle_t handle, const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	hipblasStatus_t err = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	if (err != HIPBLAS_STATUS_SUCCESS)
		std::cout << "Error: " <<  _cudaGetErrorEnum(err) << std::endl;

}


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	for(int i = 0; i < nr_rows_A; ++i){
		for(int j = 0; j < nr_cols_A; ++j){
			std::cout << A[j * nr_rows_A + i] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

int main(int argc, char* argv[]) {

	if (argc != 4){
		std::cout << "USAGE: " << argv[0] <<" <size> <inner-reps> nlinks" <<std::endl ;
		exit(-1);
	}
	int size = atoi(argv[1]);
	int reps = atoi(argv[2]);
	int nlinks = atoi(argv[3]);

	setup_nvlink();

	hipStream_t computeStream;
	hipError_t result;

	result = hipStreamCreateWithFlags(&computeStream, hipStreamNonBlocking);

	hipStream_t copyStream, copyStream2;
	hipStream_t copyStream3, copyStream4;
	result = hipStreamCreateWithFlags(&copyStream, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream2, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream3, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream4, hipStreamNonBlocking);



	// Allocate the src on CPU
	long SIZE = 512*1024*1024;
	// int* src = (int*) malloc(SIZE * sizeof(int));
	int* src; 
	int *dest_h; 
	hipHostMalloc((void**) &src, SIZE * sizeof(int), hipHostMallocDefault);
	hipHostMalloc((void**) &dest_h, SIZE * sizeof(int), hipHostMallocDefault);
	for (int i = 0; i < SIZE ; ++i) {
		src[i] = 5;
		dest_h[i] = 1;
	}

	hipSetDevice(0);
	// Allocate DST on gpu	
	int* dst;
	hipMalloc(&dst, SIZE*sizeof(int));

	// Allocate buffers on all cpus:
	hipSetDevice(1);
	int* src_1;
	hipMalloc(&src_1, SIZE*sizeof(int));

	hipMemcpy((void*)src_1, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipSetDevice(2);
	int* src_2;
	hipMalloc(&src_2, SIZE*sizeof(int));

	hipMemcpy((void*)src_2, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipSetDevice(3);
	int* src_3;
	hipMalloc(&src_3, SIZE*sizeof(int));

	hipMemcpy((void*)src_3, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();


	hipSetDevice(0);

	hipDeviceSynchronize();



	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

	// for simplicity we are going to use square arrays
	nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = size;

	float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
	float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
	float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

	// Allocate 3 arrays on GPU
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
	hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
	hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

	// If you already have useful values in A and B you can copy them in GPU:
	// cudaMemcpy(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),cudaMemcpyHostToDevice);
	// cudaMemcpy(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),cudaMemcpyHostToDevice);

	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
	GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

	// Optionally we can copy the data back on CPU and print the arrays
	hipMemcpyAsync(h_A,d_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost, computeStream);
	hipMemcpyAsync(h_B,d_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyDeviceToHost, computeStream);
	std::cout << "A =" << std::endl;
	// print_matrix(h_A, nr_rows_A, nr_cols_A);
	std::cout << "B =" << std::endl;
	// print_matrix(h_B, nr_rows_B, nr_cols_B);
	hipMemcpyAsync((void*)dst, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice, copyStream);

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetStream(handle, computeStream);

	hipDeviceSynchronize();

	for (int j = 0 ; j < 100; j++){
		// Tabkes about 5 minuets
		gpu_blas_mmul(handle, d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);
		for (int i=0; i< reps; i++){
			// each stable copy takes about 162 miliseconds
			// cudaMemcpyAsync((void*)src, (void*)dst, sizeof(int) * SIZE, cudaMemcpyDeviceToHost, copyStream);
			// cudaMemcpyAsync((void*)dest_h, (void*)dst, sizeof(int) * SIZE, cudaMemcpyDeviceToHost, copyStream2);
			if (nlinks >=1)  
				hipMemcpyAsync((void*)src_1, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToDevice, copyStream2);
			if (nlinks >= 2)
				hipMemcpyAsync((void*)src_2, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToDevice, copyStream3);
			if (nlinks >= 3)
				hipMemcpyAsync((void*)src_3, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToDevice, copyStream4);

		}
		// Create a handle for CUBLAS	        
		hipStreamSynchronize(copyStream);
		hipStreamSynchronize(copyStream2);
		hipStreamSynchronize(copyStream3);
		hipStreamSynchronize(copyStream4);

	}
	hipStreamSynchronize(computeStream);  
	hipStreamSynchronize(copyStream);
	hipStreamSynchronize(copyStream2);
	hipStreamSynchronize(copyStream3);
	hipStreamSynchronize(copyStream4);

	// Destroy the handle
	hipblasDestroy(handle);

	// Copy (and print) the result on host memory
	hipMemcpyAsync(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost, computeStream);
	std::cout << "C =" << std::endl;
	// print_matrix(h_C, nr_rows_C, nr_cols_C);

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	hipHostFree(src);
	hipHostFree(dest_h);
	hipFree(dst);	

	hipSetDevice(1);
	hipFree(src_1);

	hipSetDevice(2);
	hipFree(src_2);

	hipSetDevice(3);
	hipFree(src_3);


	result = hipStreamDestroy(computeStream);
	result = hipStreamDestroy(copyStream);
	result = hipStreamDestroy(copyStream2);
	result = hipStreamDestroy(copyStream3);
	result = hipStreamDestroy(copyStream4);

	// Free CPU memory
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
