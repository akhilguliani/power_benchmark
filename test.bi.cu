// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cmath>

// Randomization helpers 
// adapted from https://github.com/ROCmSoftwarePlatform/rocBLAS/blob/rocm-3.0/clients/include/rocblas_init.hpp#L42

void spin_loop(){
	unsigned long long loop = (unsigned long)-1;
	unsigned long long sum = 0;
	
	for (unsigned long long i=0; i < loop; i++){
		sum = sum + i;
	}
}

void init_sin(float *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = sin(i + j * nr_rows_A);
}


void init_cos(float *A, size_t nr_rows_A, size_t nr_cols_A){
    for(size_t i = 0; i < nr_rows_A; ++i)
        for(size_t j = 0; j < nr_cols_A; ++j)
	    A[i + j * nr_rows_A] = cos(i + j * nr_rows_A);
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	for(int i = 0; i < nr_rows_A; ++i){
		for(int j = 0; j < nr_cols_A; ++j){
			std::cout << A[j * nr_rows_A + i] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

// Function to enable NVLINK between card pairs 
void setup_nvlink(int numGPUs){
	if (numGPUs >= 4){
	    numGPUs = 3;
	}

	for (int i = 0; i <= numGPUs; i++) {
	    for (int j = i+1; j <= numGPUs; j++) {
		int access = 0;
		hipDeviceCanAccessPeer(&access, i, j);
		if (access) {
			printf("Enabling %d to %d\n", i, j);
			hipSetDevice(i);
			cudaCheckError();
			hipDeviceEnablePeerAccess(j, 0);
			cudaCheckError();
			hipSetDevice(j);
			cudaCheckError();
			hipDeviceEnablePeerAccess(i, 0);
			cudaCheckError();
			hipSetDevice(i);
			cudaCheckError();
		}
		fflush(stdout);
	    }
	}
}



int main(int argc, char* argv[]) {
        
	if (argc != 4){
		std::cout << "USAGE: " << argv[0] <<"<size> <reps> <active-links>" <<std::endl ;
		exit(-1);
	}

	int multiplier = atoi(argv[1]);

	int reps = atoi(argv[2]);
	
	int nlinks = atoi(argv[3]);
        
        setup_nvlink(nlinks);
	// spin_loop();

	hipSetDevice(0);
	hipStream_t copyStream, copyStream2;
	hipStream_t copyStream3, copyStream4;
	hipError_t result = hipStreamCreateWithFlags(&copyStream, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream2, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream3, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream4, hipStreamNonBlocking);

	// Allocate the src on CPU
	long SIZE = multiplier*1024*1024;
	// int* src = (int*) malloc(SIZE * sizeof(int));
	int* src; 
	hipHostMalloc((void**) &src, SIZE * sizeof(int), hipHostMallocDefault);

	int* src_h; 
	hipHostMalloc((void**) &src_h, SIZE * sizeof(int), hipHostMallocDefault);

	for (int i = 0; i < SIZE ; ++i) {
		// src[i] = (unsigned int)-1;
		src[i] = sin(i);
		src_h[i] = cos(i);
	}

	// Allocate DST on gpu	
	int* dst, dst_1, dst_2, dst_3;
	hipMalloc(&dst, SIZE*sizeof(int));

	hipMemcpyAsync((void*)dst, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice, copyStream);
	
	hipSetDevice(1);
	int* src_1;
	hipMalloc(&src_1, SIZE*sizeof(int));

	hipMemcpy((void*)src_1, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipSetDevice(2);
	int* src_2;
	hipMalloc(&src_2, SIZE*sizeof(int));

	hipMemcpy((void*)src_2, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipSetDevice(3);
	int* src_3;
	hipMalloc(&src_3, SIZE*sizeof(int));

	hipMemcpy((void*)src_3, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();


	hipSetDevice(0);

	hipDeviceSynchronize();
	// cudaMemcpyAsync((void*)src, (void*)dst, sizeof(int)*SIZE , cudaMemcpyDeviceToHost, copyStream);
        printf("START\n");
	for (int i = 0; i < reps; i++) {
            for (int j =0; j < nlinks; j++){ 
	    // cudaMemcpyAsync((void*)src, (void*)dst, sizeof(int)*SIZE , cudaMemcpyDeviceToDevice, copyStream);
	        if (j == 0) {
		   hipSetDevice(j);
		   if (nlinks >=1)  
	    	       hipMemcpyAsync((void*)src_1, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToDevice, copyStream2);
	           if (nlinks >= 2)
	    	       hipMemcpyAsync((void*)src_2, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToDevice, copyStream3);
	           if (nlinks >= 3)
	    	       hipMemcpyAsync((void*)src_3, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToDevice, copyStream4);
		}
		if (j == 1){
		   hipSetDevice(j);
		}
		
	    }        
	}

	// Create a handle for CUBLAS	        
	hipStreamSynchronize(copyStream);
	hipStreamSynchronize(copyStream2);
	hipStreamSynchronize(copyStream3);
	hipStreamSynchronize(copyStream4);

	//Free pinned memory
	hipHostFree(src);
	hipFree(dst);
	
	hipSetDevice(1);
	hipFree(src_1);
	
	hipSetDevice(2);
	hipFree(src_2);

	hipSetDevice(3);
	hipFree(src_3);


	result = hipStreamDestroy(copyStream);
	result = hipStreamDestroy(copyStream2);
	result = hipStreamDestroy(copyStream3);
	result = hipStreamDestroy(copyStream4);
	return 0;
}
