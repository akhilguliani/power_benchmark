// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <cmath>

// Randomization helpers 
void init_sin(float *A, size_t M, size_t N, size_t lda, size_t stride = 0, size_t batch_count = 1){

    for(size_t i_batch = 0; i_batch < batch_count; i_batch++)
        for(size_t i = 0; i < M; ++i)
            for(size_t j = 0; j < N; ++j)
                A[i + j * lda + i_batch * stride] = sin(i + j * lda + i_batch * stride);

}


void init_cos(float *A, size_t M, size_t N, size_t lda, size_t stride = 0, size_t batch_count = 1){

    for(size_t i_batch = 0; i_batch < batch_count; i_batch++)
        for(size_t i = 0; i < M; ++i)
            for(size_t j = 0; j < N; ++j)
                A[i + j * lda + i_batch * stride] = cos(i + j * lda + i_batch * stride);

}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	for(int i = 0; i < nr_rows_A; ++i){
		for(int j = 0; j < nr_cols_A; ++j){
			std::cout << A[j * nr_rows_A + i] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}


void setup_nvlink(){
	int numGPUs = 4;

	int i = 0;
	for (int j = 0; j < numGPUs; j++) {
		int access = 0;
		hipDeviceCanAccessPeer(&access, i, j);
		if (access) {
			printf("Enabling %d to %d\n", i, j);
			hipSetDevice(i);
			cudaCheckError();
			hipDeviceEnablePeerAccess(j, 0);
			cudaCheckError();
			hipSetDevice(j);
			cudaCheckError();
			hipDeviceEnablePeerAccess(i, 0);
			cudaCheckError();
			hipSetDevice(i);
			cudaCheckError();
		}
		fflush(stdout);
	}
}



int main(int argc, char* argv[]) {
        
	if (argc != 4){
		std::cout << "USAGE: " << argv[0] <<"<size> <reps> <active-links>" <<std::endl ;
		exit(-1);
	}

	int multiplier = atoi(argv[1]);

	int reps = atoi(argv[2]);
	
	int nlinks = atoi(argv[3]);
        
        setup_nvlink();

	hipSetDevice(0);
	hipStream_t copyStream, copyStream2;
	hipStream_t copyStream3, copyStream4;
	hipError_t result = hipStreamCreateWithFlags(&copyStream, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream2, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream3, hipStreamNonBlocking);
	result = hipStreamCreateWithFlags(&copyStream4, hipStreamNonBlocking);

	// Allocate the src on CPU
	long SIZE = multiplier*1024*1024;
	// int* src = (int*) malloc(SIZE * sizeof(int));
	int* src; 
	hipHostMalloc((void**) &src, SIZE * sizeof(int), hipHostMallocDefault);

	int* src_h; 
	hipHostMalloc((void**) &src_h, SIZE * sizeof(int), hipHostMallocDefault);

	for (int i = 0; i < SIZE ; ++i) {
		src[i] = (unsigned int)-1;
		src_h[i] = 4;
	}

	// Allocate DST on gpu	
	int* dst;
	hipMalloc(&dst, SIZE*sizeof(int));

	hipMemcpyAsync((void*)dst, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice, copyStream);
	
	hipSetDevice(1);
	int* src_1;
	hipMalloc(&src_1, SIZE*sizeof(int));

	hipMemcpy((void*)src_1, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipSetDevice(2);
	int* src_2;
	hipMalloc(&src_2, SIZE*sizeof(int));

	hipMemcpy((void*)src_2, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipSetDevice(3);
	int* src_3;
	hipMalloc(&src_3, SIZE*sizeof(int));

	hipMemcpy((void*)src_3, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice);
	hipDeviceSynchronize();


	hipSetDevice(0);

	hipDeviceSynchronize();
	
	for (int i = 0; i < reps; i++) {
	    hipMemcpyAsync((void*)src, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToHost, copyStream);
	    if (nlinks >=1)  
	    	hipMemcpyAsync((void*)src_1, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToHost, copyStream2);
	    if (nlinks >= 2)
	    	hipMemcpyAsync((void*)src_2, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToHost, copyStream3);
	    if (nlinks >= 3)
	    	hipMemcpyAsync((void*)src_3, (void*)dst, sizeof(int)*SIZE , hipMemcpyDeviceToHost, copyStream4);
        }

	// Create a handle for CUBLAS	        
	hipStreamSynchronize(copyStream);
	hipStreamSynchronize(copyStream2);
	hipStreamSynchronize(copyStream3);
	hipStreamSynchronize(copyStream4);

	//Free pinned memory
	hipHostFree(src);
	hipFree(dst);
	
	hipSetDevice(1);
	hipFree(src_1);
	
	hipSetDevice(2);
	hipFree(src_2);

	hipSetDevice(3);
	hipFree(src_3);


	result = hipStreamDestroy(copyStream);
	result = hipStreamDestroy(copyStream2);
	result = hipStreamDestroy(copyStream3);
	result = hipStreamDestroy(copyStream4);
	return 0;
}
