// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

	for(int i = 0; i < nr_rows_A; ++i){
		for(int j = 0; j < nr_cols_A; ++j){
			std::cout << A[j * nr_rows_A + i] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

int main() {

	hipStream_t copyStream;
	hipError_t result = hipStreamCreate(&copyStream);

	// Allocate the src on CPU
	long SIZE = 512*1024*1024;
	int* src = (int*) malloc(SIZE * sizeof(int));
	// int* src; 
	// cudaMallocHost((void**) &src, SIZE * sizeof(int));
	for (int i = 0; i < SIZE ; ++i) {
		src[i] = 5;
	}

	// Allocate DST on gpu	
	int* dst;
	hipMalloc(&dst, SIZE*sizeof(int));

	hipMemcpyAsync((void*)dst, (void*)src, sizeof(int)*SIZE , hipMemcpyHostToDevice, copyStream);

	// Create a handle for CUBLAS	
	hipDeviceSynchronize();

	for (int j = 0 ; j < 100; j++){
		hipMemcpyAsync((void*)src, (void*)dst, sizeof(int) * SIZE, hipMemcpyDeviceToHost, copyStream);
	} 
	hipStreamSynchronize(copyStream);

	//Free pinned memory
	hipHostFree(src);
	hipFree(dst);


	result = hipStreamDestroy(copyStream);
	return 0;
}
